#include "hip/hip_runtime.h"
#include "kernels.h"
#include "helpers.h"
#include <iostream>
#include <cmath>


__global__
void blur(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset-x)/width;
    int fsize = 5; // Filter size
    if(offset < width*height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = (offset+ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[offset*3] = output_red/hits;
        output_image[offset*3+1] = output_green/hits;
        output_image[offset*3+2] = output_blue/hits;
        }
}


void filter (unsigned char* input_image, unsigned char* output_image, int width, int height) {

    unsigned char* dev_input;
    unsigned char* dev_output;
    getError(hipMalloc( (void**) &dev_input, width*height*3*sizeof(unsigned char)));
    getError(hipMemcpy( dev_input, input_image, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice ));
 
    getError(hipMalloc( (void**) &dev_output, width*height*3*sizeof(unsigned char)));

    dim3 blockDims(512,1,1);
    dim3 gridDims((unsigned int) ceil((double)(width*height*3/blockDims.x)), 1, 1 );

    blur<<<gridDims, blockDims>>>(dev_input, dev_output, width, height); 


    getError(hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost ));

    getError(hipFree(dev_input));
    getError(hipFree(dev_output));

}

